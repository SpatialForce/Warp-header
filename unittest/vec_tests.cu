#include "hip/hip_runtime.h"
//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include <gtest/gtest.h>
#include "core/vec.h"

__global__ void add() {
    wp::vec3f a{1, 2, 3};
    wp::vec3f b{4, 5, 6};
    auto c = a + b;
}

TEST(VecCu, add) {
}